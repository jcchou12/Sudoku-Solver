#include "hip/hip_runtime.h"
// A Backtracking program in
// C++ to solve Sudoku problem
#include <iostream>
#include "timing.h"
using namespace std;
  
// UNASSIGNED is used for empty
// cells in sudoku grid
#define UNASSIGNED 0
  
#define M 16
// N is used for the size of Sudoku grid.
// Size will be NxN
// #define N 9
#define N M

#define BLOCKSIZE 1024
#define THREADS 32
  
// This function finds an entry in grid
// that is still unassigned
bool FindUnassignedLocation(int grid[N][N],
                            int& row, int& col);
  
// Checks whether it will be legal
// to assign num to the given row, col
bool isSafe(int grid[N][N], int row,
            int col, int num);
  

void possibleBoards(int* newBoards, int* old_boards, )

/* Takes a partially filled-in grid and attempts 
to assign values to all unassigned locations in 
such a way to meet the requirements for
Sudoku solution (non-duplication across rows,
columns, and boxes) */
__global__ bool SolveSudoku(int grid[N][N])
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int row, col;
  
    // If there is no unassigned location,
    // we are done
    if (!FindUnassignedLocation(grid, row, col))
        // success!
        return true;
  
    // Consider digits 1 to 9
    for (int num = 1; num <= N; num++) 
    {
          
        // Check if looks promising
        if (isSafe(grid, row, col, num)) 
        {
              
            // Make tentative assignment
            grid[row][col] = num;
  
            // Return, if success
            if (SolveSudoku(grid))
                return true;
  
            // Failure, unmake & try again
            grid[row][col] = UNASSIGNED;
        }
    }
     
    // This triggers backtracking
    return false;
}
  
/* Searches the grid to find an entry that is 
still unassigned. If found, the reference 
parameters row, col will be set the location 
that is unassigned, and true is returned. 
If no unassigned entries remain, false is returned. */
bool FindUnassignedLocation(int grid[N][N],
                            int& row, int& col)
{
    for (row = 0; row < N; row++)
        for (col = 0; col < N; col++)
            if (grid[row][col] == UNASSIGNED)
                return true;
    return false;
}
  
/* Returns a boolean which indicates whether 
an assigned entry in the specified row matches
the given number. */
bool UsedInRow(int grid[N][N], int row, int num)
{
    for (int col = 0; col < N; col++)
        if (grid[row][col] == num)
            return true;
    return false;
}
  
/* Returns a boolean which indicates whether 
an assigned entry in the specified column
matches the given number. */
bool UsedInCol(int grid[N][N], int col, int num)
{
    for (int row = 0; row < N; row++)
        if (grid[row][col] == num)
            return true;
    return false;
}
  
/* Returns a boolean which indicates whether 
an assigned entry within the specified sqrt(N)xsqrt(N) box 
matches the given number. */
bool UsedInBox(int grid[N][N], int boxStartRow,
               int boxStartCol, int num)
{
    for (int row = 0; row < sqrt(N); row++)
        for (int col = 0; col < sqrt(N); col++)
            if (grid[row + boxStartRow]
                    [col + boxStartCol] ==
                                       num)
                return true;
    return false;
}
  
/* Returns a boolean which indicates whether 
it will be legal to assign num to the given 
row, col location. */
bool isSafe(int grid[N][N], int row,
            int col, int num)
{
    /* Check if 'num' is not already placed in 
    current row, current column 
    and current sqrt(N)xsqrt(N) box */
    return !UsedInRow(grid, row, num)
           && !UsedInCol(grid, col, num)
           && !UsedInBox(grid, row - row % int(sqrt(N)),
                         col - col % int(sqrt(N)), num)
           && grid[row][col] == UNASSIGNED;
}
  
/* A utility function to print grid */
void printGrid(int grid[N][N])
{
    for (int row = 0; row < N; row++) 
    {
        for (int col = 0; col < N; col++)
            std::cout << grid[row][col] << " ";
        std::cout << endl;
    }
}
  
void checkCorrect9()
{
    // 0 means unassigned cells
    std::cout << "9x9 Tests: \n";
    std::cout << "Test 1 - solvable-random: \n";
    int grid1[N][N] = { { 3, 0, 6, 5, 0, 8, 4, 0, 0 },
                       { 5, 2, 0, 0, 0, 0, 0, 0, 0 },
                       { 0, 8, 7, 0, 0, 0, 0, 3, 1 },
                       { 0, 0, 3, 0, 1, 0, 0, 8, 0 },
                       { 9, 0, 0, 8, 6, 3, 0, 0, 5 },
                       { 0, 5, 0, 0, 9, 0, 6, 0, 0 },
                       { 1, 3, 0, 0, 0, 0, 2, 5, 0 },
                       { 0, 0, 0, 0, 0, 0, 0, 7, 4 },
                       { 0, 0, 5, 2, 0, 6, 3, 0, 0 } };
    Timer t;
    t.reset();
    if (SolveSudoku(grid1) == true)
        printGrid(grid1);
    else
        std::cout << "No solution exists";
    std::cout << "Time Taken: " << t.elapsed() << "\n";

    std::cout << "\nTest 2 - solvable-sparse: \n";
    int grid2[N][N] = { { 4, 0, 0, 0, 0, 6, 0, 0, 0 },
                       { 0, 7, 1, 0, 5, 0, 4, 0, 0 },
                       { 0, 0, 3, 0, 0, 9, 0, 0, 0 },
                       { 0, 0, 0, 4, 0, 0, 6, 0, 0 },
                       { 0, 0, 0, 0, 0, 0, 0, 2, 0 },
                       { 7, 0, 0, 0, 0, 0, 0, 1, 0 },
                       { 0, 0, 0, 0, 0, 4, 0, 0, 0 },
                       { 0, 6, 0, 0, 0, 0, 0, 8, 3 },
                       { 0, 5, 0, 1, 0, 0, 0, 0, 0} };
    t.reset();
    if (SolveSudoku(grid2) == true)
        printGrid(grid2);
    else
        std::cout << "No solution exists\n";
    std::cout << "Time Taken: " << t.elapsed() << "\n";

    std::cout << "\nTest 3 - unsolvable-random: \n";
    int grid3[N][N] = { { 1, 2, 3, 0, 0, 0, 0, 0, 0 },
                       { 0, 0, 0, 0, 4, 0, 0, 0, 0 },
                       { 0, 0, 0, 0, 0, 0, 0, 0, 4 },
                       { 0, 0, 0, 4, 0, 0, 6, 0, 0 },
                       { 0, 0, 0, 0, 0, 0, 0, 2, 0 },
                       { 7, 0, 0, 0, 0, 0, 0, 1, 0 },
                       { 0, 0, 0, 0, 0, 4, 0, 0, 0 },
                       { 0, 6, 0, 0, 0, 0, 0, 8, 3 },
                       { 0, 5, 0, 1, 0, 0, 2, 0, 0} };
    t.reset();
    if (SolveSudoku(grid3) == true)
        printGrid(grid3);
    else
        std::cout << "No solution exists\n";
    std::cout << "Time Taken: " << t.elapsed() << "\n";

    std::cout << "\nTest 4 - unsolvable-sparse: \n";
    int grid4[N][N] = { { 1, 2, 3, 0, 0, 0, 0, 0, 0 },
                       { 0, 0, 0, 0, 4, 0, 0, 0, 0 },
                       { 0, 0, 0, 0, 0, 0, 0, 0, 4 },
                       { 0, 0, 0, 0, 0, 0, 0, 0, 0 },
                       { 0, 0, 0, 0, 0, 0, 0, 0, 0 },
                       { 0, 0, 0, 0, 0, 0, 0, 0, 0 },
                       { 0, 0, 0, 0, 0, 0, 0, 0, 0 },
                       { 0, 0, 0, 0, 0, 0, 0, 0, 0 },
                       { 0, 0, 0, 0, 0, 0, 0, 0, 0 } };
    t.reset();
    if (SolveSudoku(grid4) == true)
        printGrid(grid4);
    else
        std::cout << "No solution exists \n";
    std::cout << "Time Taken: " << t.elapsed() << "\n";
}

void checkCorrect16()
{
    std::cout << "16x16 Tests: \n";
    std::cout << "Test 5 - solvable-random: \n";
    Timer t;
    int grid5[M][M] = { { 0, 0, 12, 8, 11, 0, 13, 10, 0, 2, 9, 1, 6, 15, 0, 0 },
                       { 0, 0, 0, 11, 8, 0, 0, 0, 5, 12, 13, 10, 4, 0, 0, 2 },
                       { 13, 2, 10, 0, 3, 7, 5, 0, 8, 6, 14, 15, 11, 0, 12, 16 },
                       { 16, 6, 9, 15, 2, 0, 0, 14, 11, 4, 7, 3, 5, 10, 0, 13 },
                       { 15, 12, 7, 0, 0, 0, 0, 11, 2, 0, 16, 13, 0, 4, 1, 0 },
                       { 11, 8, 6, 0, 4, 13, 3, 7, 0, 9, 0, 14, 16, 0, 0, 12 },
                       { 2, 4, 5, 9, 1, 16, 14, 6, 10, 3, 12, 11, 15, 7, 0, 0 },
                       { 14, 13, 1, 16, 15, 12, 2, 5, 0, 7, 8, 0, 9, 0, 11, 10 },
                       { 9, 1, 13, 0, 14, 11, 15, 4, 12, 16, 0, 0, 10, 6, 0, 0 },
                       { 0, 0, 4, 6, 0, 9, 0, 0, 13, 14, 0, 7, 12, 11, 0, 1 },
                       { 0, 11, 16, 12, 0, 0, 1, 0, 3, 15, 4, 6, 13, 14, 0, 7 },
                       { 0, 0, 2, 14, 0, 0, 0, 13, 9, 1, 11, 5, 8, 0, 4, 15 },
                       { 0, 7, 11, 0, 9, 2, 10, 8, 0, 0, 0, 16, 1, 12, 15, 0 },
                       { 4, 0, 15, 2, 13, 0, 11, 0, 1, 10, 6, 0, 0, 8, 16, 14 },
                       { 12, 0, 0, 1, 6, 14, 0, 15, 7, 11, 5, 2, 3, 13, 10, 9 },
                       { 10, 9, 14, 13, 7, 0, 16, 1, 4, 8, 0, 0, 0, 5, 6, 11 } };
    t.reset();
    if (SolveSudoku(grid5) == true)
        printGrid(grid5);
    else
        std::cout << "No solution exists";
    std::cout << "Time Taken: " << t.elapsed() << "\n";

    std::cout << "\nTest 2 - solvable-sparse: \n";
    int grid2[M][M] = { { 14, 0, 16, 0, 15, 0, 5, 4, 0, 0, 0, 2, 0, 0, 0, 0 },
                       { 0, 0, 0, 0, 2, 12, 7, 0, 0, 0, 0, 16, 0, 5, 0, 8 },
                       { 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 10, 5, 0, 0, 0, 0 },
                       { 0, 0, 0, 0, 0, 8, 16, 9, 7, 0, 4, 3, 0, 0, 0, 11 },
                       { 0, 0, 14, 0, 0, 15, 0, 0, 0, 0, 12, 4, 0, 0, 9, 0 },
                       { 9, 0, 0, 0, 0, 0, 8, 5, 0, 2, 0, 0, 11, 0, 0, 7 },
                       { 0, 0, 0, 4, 13, 0, 0, 6, 0, 0, 0, 0, 16, 0, 0, 0 },
                       { 0, 0, 6, 7, 0, 16, 0, 0, 0, 5, 9, 0, 0, 0, 10, 13 },
                       { 5, 0, 0, 0, 0, 0, 0, 3, 9, 0, 14, 0, 0, 0, 6, 1 },
                       { 0, 1, 0, 0, 0, 13, 0, 0, 0, 10, 0, 8, 0, 2, 0, 4 },
                       { 3, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
                       { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
                       { 0, 0, 0, 0, 0, 0, 0, 0, 14, 0, 0, 0, 0, 0, 0, 0 },
                       { 0, 0, 0, 0, 0, 2, 0, 0, 0, 0, 0, 0, 0, 3, 0, 0 },
                       { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 15 },
                       { 0, 0, 12, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0 } };
    t.reset();
    if (SolveSudoku(grid2) == true)
        printGrid(grid2);
    else
        std::cout << "No solution exists";
    std::cout << "Time Taken: " << t.elapsed() << "\n";
  
    std::cout << "\nTest 3 - unsolvable-random: \n";
    int grid3[M][M] = { { 0, 0, 12, 8, 11, 0, 13, 10, 0, 2, 9, 1, 6, 15, 0, 0 },
                       { 0, 0, 0, 11, 8, 0, 0, 0, 5, 12, 13, 10, 4, 0, 0, 2 },
                       { 13, 2, 10, 0, 3, 7, 5, 0, 8, 6, 14, 15, 11, 0, 12, 16 },
                       { 16, 6, 9, 15, 2, 0, 0, 14, 11, 4, 7, 3, 5, 10, 0, 13 },
                       { 15, 12, 7, 0, 0, 0, 0, 11, 2, 0, 16, 13, 0, 4, 1, 0 },
                       { 11, 8, 6, 0, 4, 13, 3, 7, 0, 9, 0, 14, 16, 0, 0, 12 },
                       { 2, 4, 5, 9, 1, 16, 14, 6, 10, 3, 12, 11, 15, 7, 0, 0 },
                       { 14, 13, 1, 16, 15, 12, 2, 5, 0, 7, 8, 0, 9, 0, 11, 10 },
                       { 9, 1, 13, 0, 14, 11, 15, 4, 12, 16, 0, 0, 10, 6, 0, 0 },
                       { 0, 0, 4, 6, 0, 9, 0, 0, 13, 14, 0, 7, 12, 11, 0, 1 },
                       { 0, 11, 16, 12, 0, 0, 1, 0, 3, 15, 4, 6, 13, 14, 0, 7 },
                       { 0, 0, 2, 14, 0, 0, 0, 13, 9, 1, 11, 5, 8, 0, 4, 15 },
                       { 0, 7, 11, 0, 9, 2, 10, 8, 0, 0, 0, 16, 1, 12, 15, 0 },
                       { 4, 0, 15, 2, 13, 0, 11, 0, 1, 10, 6, 0, 0, 8, 16, 14 },
                       { 12, 0, 0, 1, 6, 14, 0, 15, 7, 11, 5, 2, 3, 13, 10, 9 },
                       { 10, 9, 14, 13, 7, 0, 16, 1, 4, 8, 3, 0, 0, 5, 6, 11 } };
    t.reset();
    if (SolveSudoku(grid3) == true)
        printGrid(grid3);
    else
        std::cout << "No solution exists\n";
    std::cout << "Time Taken: " << t.elapsed() << "\n";

    std::cout << "\nTest 4 - unsolvable-sparse: \n";
    int grid4[M][M] = { { 1, 2, 3, 4, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
                       { 0, 0, 0, 0, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
                       { 0, 0, 0, 0, 0, 0, 0, 0, 0, 5, 0, 0, 0, 0, 0, 0 },
                       { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 5 },
                       { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0 },
                       { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
                       { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
                       { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
                       { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
                       { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
                       { 0, 0, 0, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
                       { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
                       { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
                       { 0, 0, 5, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
                       { 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
                       { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0 } };
    t.reset();
    if (SolveSudoku(grid4) == true)
        printGrid(grid4);
    else
        std::cout << "No solution exists\n";
    std::cout << "Time Taken: " << t.elapsed() << "\n";
}

// Driver Code
int main()
{
    // checkCorrect9();
    checkCorrect16();
    return 0;
}
